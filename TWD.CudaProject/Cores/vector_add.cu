
#include <hip/hip_runtime.h>
﻿extern "C" __global__
void VectorAdd(float* A, float* B, float* C, int N)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}